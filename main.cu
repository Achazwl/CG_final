#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <memory>

#include "config/config.h"
#include "pt/tracing.h"
#include "scene.h"
#include "utils/rnd.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void kernelRayTrace(Group *group, Camera *cam, Vec *result, hiprandState *states) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= cam->n_sub) return;
	int pixel_idx = idx / cam->subpixel2;
	int y = pixel_idx / cam->w;
	int x = pixel_idx % cam->w;
	int sy = (idx % cam->subpixel2) / cam->subpixel;
	int sx = (idx % cam->subpixel2) % cam->subpixel;

	hiprand_init(y*y*y, idx, 0, &states[idx]); // TODO: no set seed each thread
	hiprandState st = states[idx];

	Vec r = Vec();
	F cx = x + (sx+.5) / cam->subpixel, cy = y + (sy+.5) / cam->subpixel;
	for (int s = 0; s < cam->samps; s++){ 
		F dx = tent_filter(1/cam->subpixel, &st), dy = tent_filter(1/cam->subpixel, &st); // TODO: better filter (like bicubic)
		Vec d = cam->x * ( ( cx + dx ) / cam->w - 0.5) +
				cam->y * ( ( cy + dy ) / cam->h - 0.5) + 
				cam->_z; 
		r = r + tracing(group, Ray(cam->o+d*cam->length, d.normal()), &st) / cam->samps; // average over samps
	}
	result[idx] = r;
}

__global__ void kernelCombResult(Vec *subpixel, Vec *pixel, Camera *cam) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= cam->n_pixel) return;

    Vec res = Vec();
    #pragma unroll
    for (int i = 0; i < cam->subpixel2; i++) {
        Vec sub = subpixel[idx * cam->subpixel2 + i];
        res = res + Vec(clamp(sub.x), clamp(sub.y), clamp(sub.z)) / cam->subpixel2;
    }

    pixel[idx] = res;
}

__host__ int ceil_div(int x, int y) {
	return (x + y - 1) / y;
}

int main(int argc, char *argv[]) { 
	printf("initial begin\n");
	Scene scene;
	Camera *cam;
	hipMalloc((void**)&cam, sizeof(Camera));
	hipMemcpy(cam, scene.cam, sizeof(Camera), hipMemcpyHostToDevice); // cpu -> gpu
	Group *group = scene.group->to(); // cpu -> gpu
	printf("initial end\n");

	hiprandState *states;
	Vec *sub_result;
	hipMalloc((void**)&states, scene.cam->n_sub*sizeof(hiprandState));
	hipMalloc((void**)&sub_result, scene.cam->n_sub*sizeof(Vec));
	dim3 blockDim(blocksize, 1);
	dim3 gridDim(ceil_div(scene.cam->n_sub, blocksize), 1);
	kernelRayTrace<<<gridDim, blockDim>>>(group, cam, sub_result, states);
	hipPeekAtLastError();
	printf("render begin\n");
	gpuErrchk( hipDeviceSynchronize() ); // wait all
	printf("render end\n");

	Vec *pixel_result;
	hipMalloc((void**)&pixel_result, scene.cam->n_pixel*sizeof(Vec));
	dim3 gridDim2(ceil_div(scene.cam->n_pixel, blocksize), 1);
	kernelCombResult<<<gridDim2, blockDim>>>(sub_result, pixel_result, cam);
	gpuErrchk( hipDeviceSynchronize() ); // wait all
	printf("combine end\n");

	Vec *img = new RGB[scene.cam->n_pixel]; 
	hipMemcpy(img, pixel_result, scene.cam->n_pixel*sizeof(Vec), hipMemcpyDeviceToHost); // gpu to cpu
	FILE *f = fopen("image.ppm", "w"); // write to image file
	fprintf(f, "P3\n%d %d\n%d\n", scene.cam->w, scene.cam->h, 255); 
	for (int i = 0; i < scene.cam->n_pixel; i++) 
		fprintf(f,"%d %d %d ", toInt(img[i].x), toInt(img[i].y), toInt(img[i].z)); 
	printf("image output end\n");

	hipFree(states);
	hipFree(sub_result);
	hipFree(pixel_result);
	hipFree(group);
	hipFree(cam);

	return 0;
} 